#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <math.h>
using namespace std;

//*******************************************

// Write down the kernels here


//***********************************************



__global__ void clarity (int *A,int n,int val){
  int  id = blockDim.x*blockIdx.x +threadIdx.x ;
  if(id<n){
    A[id]= val ;
  }
}






__global__ void nearest(int *health ,int *kx,int *ky ,int *round,int T,int *kdistance,int *kscore , int *newhealth){


  
   int tank =blockIdx.x;
   int i = round[0];
   if (i%T!=0){
    int currenttank =   threadIdx.x;
    if (currenttank==0)
    kdistance[tank]=INT_MAX;

__syncthreads();
    int actual = (tank+i)%T ;
   int x1,x2,y1,y2,x3,y3;
   int d1 = -1,a,b,a1,b1,mycheck ;
   
if (tank!=currenttank&&health[tank]>0&&health[currenttank]>0){

    x1=kx[tank];
    y1=ky[tank];
    x2=kx[currenttank];
    y2=ky[currenttank];
    x3=kx[actual];
    y3=ky[actual];
    

     a = y3-y1;b=x3-x1;
     a1 = y2-y1 ;
     b1 = x2-x1;
    mycheck = 1 ;
  if ((x1>x2&&x1<x3)||(x1<x2&&x1>x3)||(y1>y2&&y1<y3)||(y1<y2&&y1>y3)){
            mycheck = 0;
        }
 



  //  __syncthreads();

  if((( a*b1 )== (b*a1)) && mycheck&&health[tank]>0&&health[currenttank]>0 ){
int c,d ;
c=abs(a1);
d=abs(b1);
    // d1 = c*c+d*d    ;
    d1=c+d;
        // printf("%d %d %d %d\n",tank,currenttank,d1,kdistance[tank]);
       

                atomicMin(&kdistance[tank],d1);

            }
    


    __syncthreads();
    
    if (d1==kdistance[tank]){

    
        atomicSub(&newhealth[currenttank],1);
        atomicAdd(&kscore[tank],1);

    }
}}
    }






__global__ void copypaste (int *A,int *B ){
  int id = threadIdx.x;
  A[id]=B[id];
}


__global__ void checker(int *khealth,int *check,int *kscore,int *round){
    int tankid = threadIdx.x;
    if(tankid==0) atomicAdd(&round[0],1);
    if (khealth[tankid]>0) atomicAdd(&check[0],1);
   
}

int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *kx,*ky,*khealth,*check,*kscore,*knewhealth ,*kdistance;

    int mycheck[1];
    mycheck[0]=T;// tryb pointer
   

    hipMalloc(&kx,T*(sizeof(int)));

    hipMalloc(&ky,T*(sizeof(int)));
    hipMalloc(&kdistance,T*(sizeof(int)));



    hipMalloc(&khealth,T*(sizeof(int)));
    hipMalloc(&knewhealth,T*(sizeof(int)));

    hipMalloc(&kscore,T*(sizeof(int)));

    hipMalloc(&check,(sizeof(int)));
   
        clarity<<<1,1>>>(check,1,1);

    

    hipMemcpy(kx,xcoord,T*(sizeof(int)),hipMemcpyHostToDevice);
    hipMemcpy(ky,ycoord,T*(sizeof(int)),hipMemcpyHostToDevice);

    clarity<<<1,T>>> (khealth,T,H);
    clarity<<<1,T>>> (knewhealth,T,H);
    int *round ;
    hipMalloc(&round,(sizeof(int)));
    clarity<<<1,1>>>(round,1,1);
  
 
    while (mycheck[0]>1){
     
       
       
        nearest<<<T,T>>>(khealth,kx,ky,round,T,kdistance,kscore,knewhealth);
     
       copypaste<<<1,T>>>(khealth,knewhealth);
        clarity<<<1,1>>>(check,1,0);
        checker<<<1,T>>>(khealth,check,kscore,round);
        hipDeviceSynchronize();
        hipMemcpy(mycheck,check,sizeof(int),hipMemcpyDeviceToHost);
  
        
    }


hipMemcpy(score,kscore,T*sizeof(int),hipMemcpyDeviceToHost);


    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}